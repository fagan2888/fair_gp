#include "hip/hip_runtime.h"
/* FEAT
copyright 2017 William La Cava
license: GNU/GPL v3
*/
#include "../error_handling.h"
#include "../cuda_utils.h"

namespace FT{
    namespace Pop{
        namespace Op{
                        
            __global__ void GEQ(float * xf, bool * xb, size_t idxf, size_t idxb, size_t N)
            {                    
                for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
                {
                    xb[idxb*N+i] = xf[(idxf-1)*N+i] >= xf[(idxf-2)*N+i];
                }
                return;
            }
            void GPU_GEQ(float * xf, bool * xb, size_t idxf, size_t idxb, size_t N)
            {
                GEQ<<< DIM_GRID, DIM_BLOCK >>>(xf, xb, idxf, idxb, N);
            }
            /// Evaluates the node and updates the stack states. 
            /* void NodeGEQ::evaluate(const MatrixXf& X, const VectorXf& y, vector<ArrayXf>& stack_f, */ 
            /*         vector<ArrayXb>& stack_b) */
            /* { */
            /*     ArrayXf x2 = stack_f.back(); stack_f.pop_back(); */
            /*     ArrayXf x1 = stack_f.back(); stack_f.pop_back(); */
            /*     // evaluate on the GPU */
            /*     ArrayXb result = ArrayXb(x1.size()); */
            /*     size_t N = result.size(); */
            /*     bool * dev_res; */
            /*     int numSMs; */
            /*     hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0); */
            /*     // allocate device arrays */
            /*     float * dev_x1, * dev_x2 ; */ 
            /*     HANDLE_ERROR(hipMalloc((void **)& dev_x1, sizeof(float)*N)); */
            /*     HANDLE_ERROR(hipMalloc((void **)& dev_x2, sizeof(float)*N)); */
            /*     HANDLE_ERROR(hipMalloc((void **)&dev_res, sizeof(bool)*N)); */
            /*     // Copy to device */
            /*     HANDLE_ERROR(hipMemcpy(dev_x1, x1.data(), sizeof(float)*N, hipMemcpyHostToDevice)); */
            /*     HANDLE_ERROR(hipMemcpy(dev_x2, x2.data(), sizeof(float)*N, hipMemcpyHostToDevice)); */

            /*     GEQ<<< 32*numSMs, 128 >>>(dev_x1, dev_x2, dev_res, N); */
               
            /*     // Copy to host */
            /*     HANDLE_ERROR(hipMemcpy(result.data(), dev_res, sizeof(bool)*N, hipMemcpyDeviceToHost)); */
                
            /*     stack_b.push_back(result); */
            /*     // Free memory */
            /*     hipFree(dev_x1); hipFree(dev_x2); hipFree(dev_res); */
            /* } */

        }	
    }
}


