/* FEAT
copyright 2017 William La Cava
license: GNU/GPL v3
*/
#include "state_utils.h"
#include "error_handling.h"
// stack utils

namespace FT{
    namespace Pop{
        namespace Op{

            void dev_allocate(float *& f, size_t Sizef,
                              int *& c, size_t Sizec,
                              bool *& b, size_t Sizeb)
            {
                HANDLE_ERROR(hipMalloc((void **)& f, sizeof(float)*Sizef));
                HANDLE_ERROR(hipMalloc((void **)& c, sizeof(float)*Sizec));
                HANDLE_ERROR(hipMalloc((void **)& b, sizeof(bool)*Sizeb));
	            //HANDLE_ERROR(hipDeviceSynchronize());
                //std::cout << "allocated " << sizeof(float)*Sizef << " bytes at loc " << f << " for stack.f\n";
                //std::cout << "allocated " << sizeof(bool)*Sizeb << " bytes at loc " << b << " for stack.b\n";
            }
            
            void copy_from_device(float * dev_f, float * host_f, size_t Sizef)
            {
                HANDLE_ERROR(hipMemcpy(host_f, dev_f, sizeof(float)*Sizef, hipMemcpyDeviceToHost));
            }
            
            void copy_from_device(int * dev_c, int * host_c, size_t Sizec)
            {
                HANDLE_ERROR(hipMemcpy(host_c, dev_c, sizeof(int)*Sizec, hipMemcpyDeviceToHost));
            }
            
            void copy_from_device(bool * dev_b, bool * host_b, size_t Sizeb)
            {
                HANDLE_ERROR(hipMemcpy(host_b, dev_b, sizeof(bool)*Sizeb,  hipMemcpyDeviceToHost));
            }

            void copy_from_device(float * dev_f, float * host_f, size_t Sizef,
                                  int * dev_c, int * host_c, size_t Sizec,
                                  bool * dev_b, bool * host_b, size_t Sizeb)
            {
                //std::cout << "dev_f: " << dev_f << "\nhost_f: " << host_f << "\nSizef: " << Sizef << "\nSizeb: " << Sizeb <<"\n";
                
	            copy_from_device(dev_f, host_f, Sizef);
	            copy_from_device(dev_c, host_c, Sizec);
	            copy_from_device(dev_b, host_b, Sizeb);	
	            //HANDLE_ERROR(hipDeviceSynchronize());
            }

            void free_device(float * dev_f, int * dev_c, bool * dev_b)
            {
                // Free memory
                hipFree(dev_f); 
                hipFree(dev_c);
                hipFree(dev_b);         
	            //HANDLE_ERROR(hipDeviceSynchronize());
            }
        }
    }
}
