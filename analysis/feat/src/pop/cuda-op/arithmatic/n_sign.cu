#include "hip/hip_runtime.h"
/* FEAT
copyright 2017 William La Cava
license: GNU/GPL v3
*/
#include "../error_handling.h"
#include "../cuda_utils.h"

namespace FT{
   	namespace Pop{
   	    namespace Op{	 
           	      		
            __global__ void Sign( float * x, size_t idx, size_t N)
            {                    
                for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
                {
                    float res = x[(idx-1)*N+i];
                    if (res > 0 )
                        x[(idx-1)*N+i] = 1.0 ; 
                    else if (res == 0)
                        x[(idx-1)*N+i] = 0.0; 
                    else
                        x[(idx-1)*N+i] = -1.0 ;
                }
                return;
            }
            void GPU_Sign( float * x, size_t idx, size_t N)
            {
                Sign<<< DIM_GRID, DIM_BLOCK >>>(x, idx, N);
            }
            /// Evaluates the node and updates the stack states. 
            /* void NodeSign::evaluate(const MatrixXf& X, const VectorXf& y, vector<ArrayXf>& stack_f, */ 
            /*         vector<ArrayXb>& stack_b) */
            /* { */
            /*     ArrayXf x1 = stack_f.back(); stack_f.pop_back(); */
            /*     // evaluate on the GPU */
            /*     ArrayXf result = ArrayXf(x1.size()); */
            /*     size_t N = result.size(); */
            /*     float * dev_res; */
            /*     int numSMs; */
            /*     hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0); */
            /*     // allocate device arrays */
            /*     float * dev_x1 ; */ 
            /*     HANDLE_ERROR(hipMalloc((void **)& dev_x1, sizeof(float)*N)); */
            /*     HANDLE_ERROR(hipMalloc((void **)&dev_res, sizeof(float)*N)); */
            /*     // Copy to device */
            /*     HANDLE_ERROR(hipMemcpy(dev_x1, x1.data(), sizeof(float)*N, hipMemcpyHostToDevice)); */

            /*     Sign<<< 32*numSMs, 128 >>>(dev_x1, dev_res, N); */
               
            /*     // Copy to host */
            /*     HANDLE_ERROR(hipMemcpy(result.data(), dev_res, sizeof(float)*N, hipMemcpyDeviceToHost)); */
                
            /*     stack_f.push_back(limited(result)); */
            /*     // Free memory */
            /*     hipFree(dev_x1); hipFree(dev_res); */
            /* } */

        }	
    }
}

