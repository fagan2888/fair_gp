#include "hip/hip_runtime.h"
/* FEAT
copyright 2017 William La Cava
license: GNU/GPL v3
*/
#include "../error_handling.h"
#include "../cuda_utils.h"

namespace FT{
   	namespace Pop{
   	    namespace Op{	
           		
            __global__ void Cos( float * x, size_t idx, size_t N, float W0)
            {                    
                for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
                {
                    x[(idx-1)*N+i] = cos ( W0*x[(idx-1)*N+i] );
                }
                return;
            }
            void GPU_Cos( float * x, size_t idx, size_t N, float W0)
            {
                Cos<<< DIM_GRID, DIM_BLOCK >>>(x, idx, N, W0);
            }
            /// Evaluates the node and updates the stack states. 
            /* void NodeCos::evaluate(const MatrixXf& X, const VectorXf& y, vector<ArrayXf>& stack_f, */ 
            /*         vector<ArrayXb>& stack_b) */
            /* { */
            /*     ArrayXf x1 = stack_f.back(); stack_f.pop_back(); */
            /*     // evaluate on the GPU */
            /*     ArrayXf result = ArrayXf(x1.size()); */
            /*     size_t N = result.size(); */
            /*     float * dev_res; */
            /*     int numSMs; */
            /*     hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0); */
            /*     // allocate device arrays */
            /*     float * dev_x1; */ 
            /*     HANDLE_ERROR(hipMalloc((void **)& dev_x1, sizeof(float)*N)); */
            /*     HANDLE_ERROR(hipMalloc((void **)&dev_res, sizeof(float)*N)); */
            /*     // Copy to device */
            /*     HANDLE_ERROR(hipMemcpy(dev_x1, x1.data(), sizeof(float)*N, hipMemcpyHostToDevice)); */

            /*     Cos<<< 32*numSMs, 128 >>>(dev_x1, dev_res, N); */
               
            /*     // Copy to host */
            /*     HANDLE_ERROR(hipMemcpy(result.data(), dev_res, sizeof(float)*N, hipMemcpyDeviceToHost)); */
                
            /*     stack_f.push_back(limited(result)); */
            /*     // Free memory */
            /*     hipFree(dev_x1); hipFree(dev_res); */
            /* } */

        }	
    }
}

