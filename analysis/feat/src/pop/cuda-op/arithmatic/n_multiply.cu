#include "hip/hip_runtime.h"
/* FEAT
copyright 2017 William La Cava
license: GNU/GPL v3
*/
#include "../error_handling.h"
#include "../cuda_utils.h"
#include <limits>

namespace FT{
   	namespace Pop{
   	    namespace Op{	 
           	      		
            __global__ void Multiply( float * x, size_t idx, size_t N, float W0, float W1)
            {     
            
                float MAX_FLT = std::numeric_limits<float>::max();
                float MIN_FLT = std::numeric_limits<float>::lowest();                  
               
                for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
                {
                    x[(idx-2)*N+i] = (W0*x[(idx-1)*N+i] * W1*x[(idx-2)*N+i]);
                    
                    x[(idx-2)*N+i] = (isnan(x[(idx-2)*N+i])) ? 0 : x[(idx-2)*N+i];
                    x[(idx-2)*N+i] = (x[(idx-2)*N+i] < MIN_FLT) ? MIN_FLT : x[(idx-2)*N+i];
                    x[(idx-2)*N+i] = (x[(idx-2)*N+i] > MAX_FLT) ? MAX_FLT : x[(idx-2)*N+i];
                }
                return;
            }
            void GPU_Multiply( float * x, size_t idx, size_t N, float W0, float W1)
            {
                Multiply<<< DIM_GRID, DIM_BLOCK >>>(x, idx, N, W0, W1);
            }
            /// Evaluates the node and updates the stack states. 
            /* void NodeMultiply::evaluate(const MatrixXf& X, const VectorXf& y, vector<ArrayXf>& stack_f, */ 
            /*         vector<ArrayXb>& stack_b) */
            /* { */
            /*     ArrayXf x2 = stack_f.back(); stack_f.pop_back(); */
            /*     ArrayXf x1 = stack_f.back(); stack_f.pop_back(); */
            /*     // evaluate on the GPU */
            /*     ArrayXf result = ArrayXf(x1.size()); */
            /*     size_t N = result.size(); */
            /*     float * dev_res; */
            /*     int numSMs; */
            /*     hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0); */
            /*     // allocate device arrays */
            /*     float * dev_x1, * dev_x2 ; */ 
            /*     HANDLE_ERROR(hipMalloc((void **)& dev_x1, sizeof(float)*N)); */
            /*     HANDLE_ERROR(hipMalloc((void **)& dev_x2, sizeof(float)*N)); */
            /*     HANDLE_ERROR(hipMalloc((void **)&dev_res, sizeof(float)*N)); */
            /*     // Copy to device */
            /*     HANDLE_ERROR(hipMemcpy(dev_x1, x1.data(), sizeof(float)*N, hipMemcpyHostToDevice)); */
            /*     HANDLE_ERROR(hipMemcpy(dev_x2, x2.data(), sizeof(float)*N, hipMemcpyHostToDevice)); */

            /*     Multiply<<< 32*numSMs, 128 >>>(dev_x1, dev_x2, dev_res, N); */
               
            /*     // Copy to host */
            /*     HANDLE_ERROR(hipMemcpy(result.data(), dev_res, sizeof(float)*N, hipMemcpyDeviceToHost)); */
                
            /*     stack_f.push_back(limited(result)); */
            /*     // Free memory */
            /*     hipFree(dev_x1); hipFree(dev_x2); hipFree(dev_res); */
            /* } */

        }	
    }
}

