#include "hip/hip_runtime.h"
/* FEAT
copyright 2017 William La Cava
license: GNU/GPL v3
*/
#include "../error_handling.h"
#include "../cuda_utils.h"

namespace FT{
   	namespace Pop{
   	    namespace Op{	
           	       		
            __global__ void Float(float * x, bool* y, size_t idxf, size_t idxb, size_t N)
            {                    
	            for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
                    x[(idxf)*N + i] = float(y[(idxb-1)*N + i]);
                    
                return;
            }
            
            __global__ void Float(float * x, int* y, size_t idxf, size_t idxi, size_t N)
            {                    
	            for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
                    x[(idxf)*N + i] = float(y[(idxi-1)*N + i]);
                    
                return;
            }
            
            void GPU_Float(float * x, bool* y, size_t idxf, size_t idxb, size_t N)
            {
                Float<<< DIM_GRID, DIM_BLOCK >>>(x, y, idxf, idxb, N);
            }
            
            void GPU_Float(float * x, int* y, size_t idxf, size_t idxi, size_t N)
            {
                Float<<< DIM_GRID, DIM_BLOCK >>>(x, y, idxf, idxi, N);
            }
        }	
    }
}


