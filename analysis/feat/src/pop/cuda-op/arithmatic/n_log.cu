#include "hip/hip_runtime.h"
/* FEAT
copyright 2017 William La Cava
license: GNU/GPL v3
*/
#include "../error_handling.h"
#include "../cuda_utils.h"
#include <limits>

namespace FT{
   	namespace Pop{
   	    namespace Op{	 
           	      		
            __global__ void Log(float * x, size_t idx, size_t N, float W0)
            {                    
                for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
                {
                    if ( abs(x[(idx-1)*N+i]) > 0.000000001) 
                        x[(idx-1)*N+i] = log(abs(W0*x[(idx-1)*N+i]));
                    else
                        x[(idx-1)*N+i] = std::numeric_limits<float>::lowest(); 
                }
                return;
            }
            void GPU_Log(float * x, size_t idx, size_t N, float W0)
            {
                Log<<< DIM_GRID, DIM_BLOCK >>>(x, idx, N, W0);
            }
            /// Evaluates the node and updates the stack states. 
            /* void NodeLog::evaluate(const MatrixXf& X, const VectorXf& y, vector<ArrayXf>& stack_f, */ 
            /*         vector<ArrayXb>& stack_b) */
            /* { */
            /*     ArrayXf x1 = stack_f.back(); stack_f.pop_back(); */
            /*     // evaluate on the GPU */
            /*     ArrayXf result = ArrayXf(x1.size()); */
            /*     size_t N = result.size(); */
            /*     float * dev_res; */
            /*     int numSMs; */
            /*     hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0); */
            /*     // allocate device arrays */
            /*     float * dev_x1  ; */ 
            /*     HANDLE_ERROR(hipMalloc((void **)& dev_x1, sizeof(float)*N)); */
            /*     HANDLE_ERROR(hipMalloc((void **)&dev_res, sizeof(float)*N)); */
            /*     // Copy to device */
            /*     HANDLE_ERROR(hipMemcpy(dev_x1, x1.data(), sizeof(float)*N, hipMemcpyHostToDevice)); */

            /*     Log<<< 32*numSMs, 128 >>>(dev_x1, dev_res, N); */
               
            /*     // Copy to host */
            /*     HANDLE_ERROR(hipMemcpy(result.data(), dev_res, sizeof(float)*N, hipMemcpyDeviceToHost)); */
                
            /*     stack_f.push_back(limited(result)); */
            /*     // Free memory */
            /*     hipFree(dev_x1); hipFree(dev_res); */
            /* } */

        }	
    }
}

